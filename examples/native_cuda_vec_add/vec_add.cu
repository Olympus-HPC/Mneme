#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

template <typename T> __global__ 
void vecAdd(T *in, T *out, size_t size) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= size)
    return;
  auto stride = gridDim.x * blockDim.x;

  for (; tid < size; tid += stride) {
    out[tid] += in[tid] + tid;
  }
}

int main(int argc, const char *argv[]) {
  size_t numElements = atoi(argv[1]);
  double *in, *out;
  double val = numElements;
  hipMalloc((void **)&in, numElements * sizeof(double));
  hipMalloc((void **)&out, numElements * sizeof(double));
  std::cout << "In : " << in << " Out " << out << "\n";

  for (int i = 0; i < 10 ; i++){
    hipMemset(in, 0, numElements * sizeof(double));
    hipMemset(out, 0, numElements * sizeof(double));

    const int threads = 256;
    int num_blocks = (numElements + threads - 1) / threads;
    vecAdd<<< num_blocks, threads>>>(in, out, val, numElements);
    hipDeviceSynchronize();
  }

  double *h_in = new double[numElements];
  double *h_out = new double[numElements];
  hipMemcpy(h_in, in, sizeof(double)*numElements, hipMemcpyDeviceToHost);
  hipMemcpy(h_out, out, sizeof(double)*numElements, hipMemcpyDeviceToHost);
  for (int i = 0; i < numElements; i++){
    if (h_in[i] + i != h_out[i]){
      std::cout << "Values at " << i << " differ\n";
      std::cout << "Values " << h_in[i] << " " << h_out[i] << "differ\n";
      return -1;
    }
  }
  
  delete [] h_in;
  delete [] h_out;
  hipFree(in);
  hipFree(out);
  return 0;
}
